#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <WinSock2.h>
#include <WS2tcpip.h>
#include "mpi.h"

#define DEF_BUFLEN 32
#define AES_PORT "21522"
#define MEGABYTE_SIZE 1024*1024
#define CUDA_BLOCKS 65536
#define CUDA_THREADS_PER_BLOCK 512

typedef uint8_t aesbyte_t;

struct AESMatrix {
	/* ������� ������������ �������� ����. */
	aesbyte_t* grid;
	aesbyte_t side_len;
};

/* ������� ������������� �������. */
void aes_matrix_init(struct AESMatrix* m, aesbyte_t side_len) {
	/* ��������� ����� ������� �� ������������. */
	if (!side_len) {
		fprintf(stderr, "ERROR in aes_matrix_init(): wrong side length specified \"%d\".\n", side_len);
		return;
	}
	/* ������ ����� ������� �������, �������� ������ � ��������� ���������. */
	m->side_len = side_len;
	m->grid = (aesbyte_t*)malloc(sizeof(aesbyte_t) * side_len * side_len);
	if (!m->grid) {
		fprintf(stderr, "ERROR in aes_matrix_init(): couldn't allocate memory for matrix grid pointer.\n");
		return;
	}
}

/* ������� �������� �������. */
void aes_matrix_free(struct AESMatrix* m) {
	/* ��������� ���������. */
	if (!m) {
		fprintf(stderr, "ERROR in aes_matrix_free(): got uninitialised matrix.\n");
		return;
	}
	if (!m->grid)
		fprintf(stderr, "WARNING in aes_matrix_free(): got null pointer.\n");
	/* ���������� � ��������� ������ �������, � ��������� �� ���� �������. */
	else
		free(m->grid);
}

/* ������� ������ ������� � ����������� �����. */
void aes_matrix_display(struct AESMatrix m) {
	/* ��������� ���������. */
	if (!m.grid) {
		fprintf(stderr, "ERROR in aes_matrix_display(): got uninitialised matrix.\n");
		return;
	}
	/* ��������� ����� ������� �������. */
	if (!m.side_len) {
		fprintf(stderr, "ERROR in aes_matrix_display(): got matrix with null size.\n");
		return;
	}
	/* ������� ������� ��������� � ����������� �����. */
	for (aesbyte_t i = 0; i < m.side_len; i += 1) {
		for (aesbyte_t j = 0; j < m.side_len; j += 1)
			/* �� ������� ������ �������� �� ������, ���� ����� ��������� � ����. */
			if (j != m.side_len - 1)
				fprintf(stdout, "%3d   ", m.grid[i * m.side_len + j]);
			else
				fprintf(stdout, "%3d", m.grid[i * m.side_len + j]);
		fprintf(stdout, "\n");
	}
}

/* ������� ���������� ������� ���������� �������. */
void aes_matrix_randomise(struct AESMatrix m) {
	/* ��������� ���������. */
	if (!m.grid) {
		fprintf(stderr, "ERROR in aes_matrix_randomise(): got uninitialised matrix.\n");
		return;
	}
	/* ��������� ����� ������� �������. */
	if (!m.side_len) {
		fprintf(stderr, "ERROR in aes_matrix_randomise(): got matrix with null size.\n");
		return;
	}
	/* ��������� ������� ���������� �������. */
	for (aesbyte_t i = 0; i < m.side_len * m.side_len; i += 1)
		m.grid[i] = (aesbyte_t)rand();
}

/* ������� �������� ShiftRows �� AES. */
void aes_matrix_shift_rows(struct AESMatrix m) {
	/* ��������� ���������. */
	if (!m.grid) {
		fprintf(stderr, "ERROR in aes_matrix_shift_rows(): got uninitialised matrix.\n");
		return;
	}
	/* ��������� ����� ������� �������. */
	if (!m.side_len) {
		fprintf(stderr, "ERROR in aes_matrix_shift_rows(): got matrix with null size.\n");
		return;
	}
	/* �������� ������� ����� ���������, �� ����� ���������,
	   ������ ������� ������, ������� � ����. */
	aesbyte_t* temp_row = (aesbyte_t*)malloc(sizeof(aesbyte_t) * m.side_len);
	if (!temp_row) {
		fprintf(stderr, "ERROR in aes_matrix_shift_rows(): couldn't allocate memory for temp row.\n");
		return;
	}
	for (aesbyte_t i = 0; i < m.side_len; i += 1) {
		for (aesbyte_t j = 0; j < m.side_len; j += 1)
			temp_row[j] = m.grid[i * m.side_len + j];
		for (aesbyte_t j = 0; j < m.side_len; j += 1)
			m.grid[i * m.side_len + j] = temp_row[(i + j) % m.side_len];
	}
	free(temp_row);
}

int tcp_establish_conn_client(SOCKET* connect_socket) {
	/* ������ ���� �� localhost. */
	const char* server_addr = "127.0.0.1";
	/* WSADATA �������� �������� � ���������� ������� � Windows. */
	WSADATA wsa_data;
	/* ��������� addrinfo �������� ������ �� ������� � ����������. */
	struct addrinfo* result = NULL;
	struct addrinfo hints;
	int my_err;
	/* WSAStartup ����������� ������ WinSocket 2.2 � �� ��� ��������
	   ������������ ������ ����. */
	fprintf(stdout, "INFO: initialising WinSocket2 on client.\n");
	my_err = WSAStartup(MAKEWORD(2, 2), &wsa_data);
	if (my_err) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): couldn't initialise WinSocket2, code %d.\n", my_err);
		WSACleanup();
		return -1;
	}
	/* ������ ���������� �� ��������� TCP ����� �����. */
	memset(&hints, 0, sizeof(hints));
	hints.ai_family = AF_UNSPEC;
	hints.ai_socktype = SOCK_STREAM;
	hints.ai_protocol = IPPROTO_TCP;
	/* ��������� ����� ������� � ������. */
	fprintf(stdout, "INFO: resolving server address.\n");
	my_err = getaddrinfo(server_addr, AES_PORT, &hints, &result);
	if (my_err != 0) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): couldn't resolve server address, code %d.\n", my_err);
		WSACleanup();
		return -1;
	}
	*connect_socket = INVALID_SOCKET;
	/* ���������� ������������ �� ������, ����������� ����� getaddrinfo, ������� �����. */
	fprintf(stdout, "INFO: creating connect socket on client.\n");
	*connect_socket = socket(result->ai_family, result->ai_socktype, result->ai_protocol);
	if (*connect_socket == INVALID_SOCKET) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): couldn't create connect socket, code %d.\n", WSAGetLastError());
		freeaddrinfo(result);
		WSACleanup();
		return -1;
	}
	/* ������������ � �������. */
	fprintf(stdout, "INFO: connecting to server.\n");
	my_err = connect(*connect_socket, result->ai_addr, (int)result->ai_addrlen);
	if (my_err == SOCKET_ERROR) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): couldn't connect to server, code %d.\n", WSAGetLastError());
		closesocket(*connect_socket);
		freeaddrinfo(result);
		WSACleanup();
		return -1;
	}
	freeaddrinfo(result);
	if (*connect_socket == INVALID_SOCKET) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): unable to connect to server, code %d.\n", WSAGetLastError());
		WSACleanup();
		return -1;
	}
	return 0;
}

char* tcp_receive_matrices(size_t* recv_datasize, SOCKET connect_socket) {
	/* ������ ��� ����� �������������� � ������� ������,
	   � ����� ��� ����� ������. */
	char tcp_datasize_recvbuf[DEF_BUFLEN];
	char* tcp_aes_data_recvbuf;
	size_t tcp_datasize;
	int recv_res;
	/* ������� �������������� � ������� ������������ ������ � ��������� � �������� ������. */
	memset(tcp_datasize_recvbuf, 0, DEF_BUFLEN);
	recv_res = recv(connect_socket, tcp_datasize_recvbuf, DEF_BUFLEN, 0);
	if (recv_res == 0) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): connection to server closed prematurely.\n");
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	else if (recv_res < 0) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): reception failure, code %d.\n", WSAGetLastError());
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	else if (recv_res == DEF_BUFLEN) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): buffer overflow while receiving %d bytes.\n", recv_res);
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	fprintf(stdout, "INFO: client received metainfo: %d bytes, buffer is \"%s\".\n", recv_res, tcp_datasize_recvbuf);
	tcp_datasize = atoi(tcp_datasize_recvbuf);
	if (!tcp_datasize) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): received invalid data size \"%s\".\n", tcp_datasize_recvbuf);
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	/* ������� ������� �� �������. */
	tcp_aes_data_recvbuf = (char*)malloc(tcp_datasize);
	if (!tcp_aes_data_recvbuf) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): couldn't allocate reception buffer memory.\n");
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	recv_res = recv(connect_socket, tcp_aes_data_recvbuf, (int)tcp_datasize, 0);
	if (recv_res == 0) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): connection to server closed prematurely.\n");
		free(tcp_aes_data_recvbuf);
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	else if (recv_res < 0) {
		fprintf(stderr, "ERROR in tcp_receive_matrices(): reception failure, code %d.\n", WSAGetLastError());
		free(tcp_aes_data_recvbuf);
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	else if (recv_res != tcp_datasize) {
		free(tcp_aes_data_recvbuf);
		fprintf(stderr, "ERROR in tcp_receive_matrices(): client received %d bytes when %llu were specified.\n", recv_res, tcp_datasize);
		closesocket(connect_socket);
		WSACleanup();
		return (char*)NULL;
	}
	*recv_datasize = recv_res;
	fprintf(stdout, "INFO: client received matrices: %d bytes.\n", recv_res);
	fprintf(stdout, "INFO: buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", tcp_aes_data_recvbuf[0], tcp_aes_data_recvbuf[1], tcp_aes_data_recvbuf[2], tcp_aes_data_recvbuf[3], tcp_aes_data_recvbuf[tcp_datasize - 4], tcp_aes_data_recvbuf[tcp_datasize - 3], tcp_aes_data_recvbuf[tcp_datasize - 2], tcp_aes_data_recvbuf[tcp_datasize - 1]);
	return tcp_aes_data_recvbuf;
}

struct AESMatrix* aes_unflatten_matrices(char* aes_matrices_flat, size_t aes_datasize, size_t* matrix_qty) {
	fprintf(stdout, "INFO: unflattening matrices.\n");
	/* ��������� ���������. */
	if (!aes_matrices_flat) {
		fprintf(stderr, "ERROR in aes_unflatten_matrices(): got null pointer.\n");
		return (struct AESMatrix*)NULL;
	}
	aesbyte_t aes_side_len = (aesbyte_t)aes_matrices_flat[0];
	size_t aes_matrix_size = (size_t)aes_side_len * (size_t)aes_side_len + 1;
	size_t aes_matrix_qty = aes_datasize / aes_matrix_size;
	fprintf(stdout, "INFO: matrix size is %llu bytes, matrix side length is %d.\n", aes_matrix_size, aes_side_len);
	*matrix_qty = aes_matrix_qty;
	/* ���������� ����� ������, � ������� ����������� �����. */
	fprintf(stdout, "INFO: allocating memory for %llu matrices.\n", aes_matrix_qty);
	struct AESMatrix* aes_matrices = (struct AESMatrix*)malloc(sizeof(struct AESMatrix) * aes_matrix_qty);
	if (!aes_matrices) {
		fprintf(stderr, "ERROR in aes_unflatten_matrices(): couldn't allocate memory for unflattened matrices.\n");
		return (struct AESMatrix*)NULL;
	}
	fprintf(stdout, "INFO: writing matrix structs.\n");
	for (size_t i = 0; i < aes_matrix_qty; i += 1) {
		aes_matrix_init(&(aes_matrices[i]), aes_side_len);
		for (size_t j = 0; j <= aes_matrix_size; j += 1)
			aes_matrices[i].grid[j] = aes_matrices_flat[i * aes_matrix_size + j + 1];
	}
	fprintf(stdout, "INFO: displaying first matrix -\n");
	aes_matrix_display(aes_matrices[0]);
	fprintf(stdout, "INFO: displaying last matrix -\n");
	aes_matrix_display(aes_matrices[aes_matrix_qty - 1]);
	return aes_matrices;
}

int tcp_close_conn_server(SOCKET client_socket) {
	int my_err;
	/* ������� ����������. */
	fprintf(stdout, "INFO: shutting down connection on server.\n");
	my_err = shutdown(client_socket, SD_SEND);
	if (my_err == SOCKET_ERROR) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): shutting down connection on server failed with code %d.\n", WSAGetLastError());
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	closesocket(client_socket);
	WSACleanup();
	return 0;
}

char* aes_pack_matrices(struct AESMatrix* gen_matrices, size_t gen_len, size_t gen_matrix_size) {
	if (!gen_matrices) {
		fprintf(stderr, "ERROR in aes_pack_matrices(): got null pointer.\n");
		return (char*)NULL;
	}
	fprintf(stdout, "INFO: packing matrices into TCP buffer format.\n");
	size_t gen_real_size = gen_len * gen_matrix_size;
	char* gen_chars = (char*)malloc(gen_real_size * sizeof(char));
	if (!gen_chars) {
		fprintf(stderr, "ERROR in aes_pack_matrices(): couldn't allocate memory for flattened data.\n");
		return (char*)NULL;
	}
	/* ��������� ������� � ������ ����� ��� �������� �� TCP. */
	for (size_t i = 0; i < gen_len; i += 1) {
		gen_chars[i * gen_matrix_size] = gen_matrices[i].side_len;
		for (aesbyte_t j = 1; j <= gen_matrices[i].side_len * gen_matrices[i].side_len; j += 1) {
			gen_chars[(i * gen_matrix_size) + j] = gen_matrices[i].grid[j - 1];
		}
	}
	fprintf(stdout, "INFO: buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", gen_chars[0], gen_chars[1], gen_chars[2], gen_chars[3], gen_chars[gen_real_size - 4], gen_chars[gen_real_size - 3], gen_chars[gen_real_size - 2], gen_chars[gen_real_size - 1]);
	return gen_chars;
}

int tcp_establish_conn_server(SOCKET* cl_socket) {
	/* WSADATA �������� �������� � ���������� ������� � Windows. */
	WSADATA wsa_data;
	int my_err;
	/* ��������� addrinfo �������� ������ �� ������� � ����������. */
	struct addrinfo* result = NULL;
	struct addrinfo hints;
	/* WSAStartup ����������� ������ WinSocket 2.2 � �� ��� ��������
	   ������������ ������ ����. */
	fprintf(stdout, "INFO: initialising WinSocket2 on server.\n");
	my_err = WSAStartup(MAKEWORD(2, 2), &wsa_data);
	if (my_err) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): couldn't initialise WinSocket2, code %d.\n", my_err);
		WSACleanup();
		return -1;
	}
	memset(&hints, 0, sizeof(hints));
	/* ������ ���������� �� ��������� TCP ����� �����. */
	hints.ai_family = AF_INET;
	hints.ai_socktype = SOCK_STREAM;
	hints.ai_protocol = IPPROTO_TCP;
	hints.ai_flags = AI_PASSIVE;
	/* ��������� ����������� ����� � ���� �������. */
	fprintf(stdout, "INFO: resolving local address and port on server.\n");
	my_err = getaddrinfo(NULL, AES_PORT, &hints, &result);
	if (my_err) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): resolving local address and port on server failed with code %d.\n", my_err);
		WSACleanup();
		return -1;
	}
	/* ������� ����� ��� �������������. */
	fprintf(stdout, "INFO: opening listening socket on server.\n");
	SOCKET listening_socket = INVALID_SOCKET;
	listening_socket = socket(result->ai_family, result->ai_socktype, result->ai_protocol);
	if (listening_socket == INVALID_SOCKET) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): opening listening socket on server failed with code %d.\n", WSAGetLastError());
		freeaddrinfo(result);
		WSACleanup();
		return -1;
	}
	/* ��������� �����. */
	fprintf(stdout, "INFO: binding listening socket.\n");
	my_err = bind(listening_socket, result->ai_addr, (int)result->ai_addrlen);
	if (my_err == SOCKET_ERROR) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): binding socket failed with code %d.\n", WSAGetLastError());
		freeaddrinfo(result);
		closesocket(listening_socket);
		WSACleanup();
		return -1;
	}
	freeaddrinfo(result);
	/* �����, ���� �� ����������� ������. */
	fprintf(stdout, "INFO: server listening socket.\n");
	if (listen(listening_socket, SOMAXCONN) == SOCKET_ERROR) {
		printf("ERROR in tcp_send_matrices(): listening socket on server failed with code %d.\n", WSAGetLastError());
		closesocket(listening_socket);
		WSACleanup();
		return -1;
	}
	SOCKET client_socket;
	client_socket = INVALID_SOCKET;
	/* ������� ����������� ������� ��� ��� ���������. */
	client_socket = accept(listening_socket, NULL, NULL);
	if (client_socket == INVALID_SOCKET) {
		printf("ERROR in tcp_send_matrices(): accepting client connection failed with code %d.\n", WSAGetLastError());
		closesocket(listening_socket);
		WSACleanup();
		return -1;
	}
	*cl_socket = client_socket;
	return 0;
}

int tcp_send_matrices(char* aes_data, size_t aes_datasize, SOCKET client_socket) {
	/* ������ ��� ����� �������������� � ������� ������,
		� ����� ��� ����� ������. */
	char tcp_datasize_buf[DEF_BUFLEN];
	char* tcp_aes_data_sendbuf = aes_data;
	int my_err;
	if (!_itoa((int)aes_datasize, tcp_datasize_buf, 10)) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): conversion of data size %llu failed.\n", aes_datasize);
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	/* ��������� ������� �������������� � ������� ������������ ������ � ���������. */
	my_err = send(client_socket, tcp_datasize_buf, (int)strlen(tcp_datasize_buf), 0);
	if (my_err != SOCKET_ERROR)
		fprintf(stdout, "INFO: server sent metainfo to client: %d bytes, buffer is \"%s\".\n", my_err, tcp_datasize_buf);
	else {
		fprintf(stderr, "ERROR in tcp_send_matrices(): send failed with code %d.\n", WSAGetLastError());
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	/* ��������� ������� �������. */
	my_err = send(client_socket, tcp_aes_data_sendbuf, (int)aes_datasize, 0);
	if (my_err != SOCKET_ERROR)
		fprintf(stdout, "INFO: server sent matrices to client: %d bytes.\n", my_err);
	else {
		fprintf(stderr, "ERROR in tcp_send_matrices(): send failed with code %d.\n", WSAGetLastError());
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	return 0;
}

int tcp_send_time(int aes_time, SOCKET client_socket) {
	/* ����� ��� �������� ���������� � ������� ����������. */
	char tcp_time_buf[DEF_BUFLEN];
	int my_err;
	if (!_itoa(aes_time, tcp_time_buf, 10)) {
		fprintf(stderr, "ERROR in tcp_send_matrices(): conversion of time %d failed.\n", aes_time);
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	/* ��������� ������� �������������� � ������� ����������. */
	my_err = send(client_socket, tcp_time_buf, (int)strlen(tcp_time_buf), 0);
	if (my_err != SOCKET_ERROR)
		fprintf(stdout, "INFO: sent time info: %d bytes, buffer is \"%s\".\n", my_err, tcp_time_buf);
	else {
		fprintf(stderr, "ERROR in tcp_send_time(): send failed with code %d.\n", WSAGetLastError());
		closesocket(client_socket);
		WSACleanup();
		return -1;
	}
	return 0;
}

int aes_shiftrows_serial(struct AESMatrix* m_arr, size_t matrix_qty) {
	fprintf(stdout, "INFO: performing serial AES shift rows on matrix array.\n");
	if (!m_arr) {
		fprintf(stderr, "ERROR in aes_shiftrows_serial(): got null pointer.\n");
		return -1;
	}
	clock_t start = clock();
	aesbyte_t* temp_row = (aesbyte_t*)malloc(sizeof(aesbyte_t) * m_arr[0].side_len);
	for (size_t k = 0; k < matrix_qty; k += 1) {
		for (aesbyte_t i = 0; i < m_arr[k].side_len; i += 1) {
			for (aesbyte_t j = 0; j < m_arr[k].side_len; j += 1)
				temp_row[j] = m_arr[k].grid[i * m_arr[k].side_len + j];
			for (aesbyte_t j = 0; j < m_arr[k].side_len; j += 1)
				m_arr[k].grid[i * m_arr[k].side_len + j] = temp_row[(i + j) % m_arr[k].side_len];
		}
	}
	free(temp_row);
	clock_t end = (clock() - start) / (CLOCKS_PER_SEC / 1000);
	fprintf(stdout, "INFO: AES shift rows on matrix array completed.\n");
	fprintf(stdout, "INFO: displaying first matrix -\n");
	aes_matrix_display(m_arr[0]);
	fprintf(stdout, "INFO: displaying last matrix -\n");
	aes_matrix_display(m_arr[matrix_qty - 1]);
	fprintf(stdout, "INFO: computations took %d ms.\n", end);
	return (int)end;
}

__global__ void aes_shift_matrices_cuda(char* aes_matrices, size_t aes_matrix_size, aesbyte_t aes_side_len, size_t aes_matrix_qty) {
	size_t my_index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t offset;
	char* temp_row;
	char* temp_mat;
	if (my_index < aes_matrix_qty) {
		temp_mat = (char*)malloc(sizeof(aesbyte_t) * aes_matrix_size);
		temp_row = (char*)malloc(sizeof(aesbyte_t) * aes_side_len);
		offset = aes_matrix_size * my_index + 1;
		memcpy(temp_mat, aes_matrices + offset, aes_matrix_size);
		for (aesbyte_t i = 0; i < aes_side_len; i += 1) {
			for (aesbyte_t j = 0; j < aes_side_len; j += 1)
				temp_row[j] = temp_mat[i * aes_side_len + j];
			for (aesbyte_t j = 0; j < aes_side_len; j += 1)
				temp_mat[i * aes_side_len + j] = temp_row[(i + j) % aes_side_len];
		}
		memcpy(aes_matrices + offset, temp_mat, aes_matrix_size);
		free(temp_row);
		free(temp_mat);
	}
}

int aes_shiftrows_cuda_mpi(char* aes_matrices_flat, size_t aes_datasize, size_t* matrix_qty) {
	char* cuda_matrices;
	fprintf(stdout, "INFO: unflattening matrices.\n");
	/* ��������� ���������. */
	if (!aes_matrices_flat) {
		fprintf(stderr, "ERROR in aes_unflatten_matrices(): got null pointer.\n");
		return -1;
	}
	aesbyte_t aes_side_len = (aesbyte_t)aes_matrices_flat[0];
	size_t launch_blocks, launch_threads;
	size_t aes_matrix_size = (size_t)aes_side_len * (size_t)aes_side_len + 1;
	size_t aes_matrix_qty = aes_datasize / aes_matrix_size;
	fprintf(stdout, "INFO: matrix size is %llu bytes, matrix side length is %d.\n", aes_matrix_size, aes_side_len);
	*matrix_qty = aes_matrix_qty;
	fprintf(stdout, "INFO: Received %llu AES matrices.\n", aes_matrix_qty);
	fprintf(stdout, "INFO: buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", aes_matrices_flat[0], aes_matrices_flat[1], aes_matrices_flat[2], aes_matrices_flat[3], aes_matrices_flat[aes_datasize - 4], aes_matrices_flat[aes_datasize - 3], aes_matrices_flat[aes_datasize - 2], aes_matrices_flat[aes_datasize - 1]);
	fprintf(stdout, "INFO: ID0 - creating a contiguous MPI data type for metadata.\n");
	clock_t start = clock(), end;
	MPI_Datatype MPI_AES_META;
	MPI_Type_contiguous(2, MPI_UNSIGNED_LONG_LONG, &MPI_AES_META);
	MPI_Type_commit(&MPI_AES_META);
	size_t aes_meta[2];
	aes_meta[0] = aes_matrix_size;
	fprintf(stdout, "INFO: ID0 - async-broadcasting matrix size and quantity metadata.\n");
	MPI_Request req[7];
	int comm_size;
	MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
	size_t portion = aes_matrix_qty / comm_size;
	size_t last_portion = aes_matrix_qty - portion * (comm_size - 1);
	for (int i = 1; i < comm_size; i += 1) {
		aes_meta[1] = (i == comm_size - 1) ? last_portion : portion;
		MPI_Isend(aes_meta, 1, MPI_AES_META, i, 0, MPI_COMM_WORLD, &(req[i - 1]));
	}
	MPI_Barrier(MPI_COMM_WORLD);
	fprintf(stdout, "INFO: ID0 - creating a contiguous MPI data type for AES matrix.\n");
	MPI_Datatype MPI_AES_MATRIX;
	MPI_Type_contiguous((int)aes_matrix_size, MPI_CHAR, &MPI_AES_MATRIX);
	MPI_Type_commit(&MPI_AES_MATRIX);
	fprintf(stdout, "INFO: ID0 - async-broadcasting matrix array portions.\n");
	for (int i = 1; i < comm_size; i += 1) {
		MPI_Isend(aes_matrices_flat + (portion * i * aes_matrix_size), (i == comm_size - 1) ? last_portion : portion, MPI_AES_MATRIX, i, 0, MPI_COMM_WORLD, &(req[i - 1]));
	}
	fprintf(stdout, "INFO: ID0 - performing AES shift on my portion.\n");
	fprintf(stdout, "INFO: ID0 - allocating memory on CUDA device.\n");
	hipMalloc((void**)&cuda_matrices, portion * aes_matrix_size);
	fprintf(stdout, "INFO: ID0 - copying AES matrices to CUDA device.\n");
	hipMemcpy(cuda_matrices, aes_matrices_flat, portion * aes_matrix_size, hipMemcpyHostToDevice);
	launch_blocks = portion / CUDA_THREADS_PER_BLOCK + 1;
	launch_threads = CUDA_THREADS_PER_BLOCK;
	fprintf(stdout, "INFO: ID0 - launching CUDA kernel on device: %llu blocks, %llu threads.\n", launch_blocks, launch_threads);
	aes_shift_matrices_cuda <<<launch_blocks, launch_threads>>>(cuda_matrices, aes_matrix_size, aes_side_len, portion);
	fprintf(stdout, "INFO: ID0 - AES shift rows on matrix array completed.\n");
	fprintf(stdout, "INFO: ID0 - fetching shifted AES matrices from CUDA device.\n");
	hipMemcpy(aes_matrices_flat, cuda_matrices, portion * aes_matrix_size, hipMemcpyDeviceToHost);
	end = (clock() - start) / (CLOCKS_PER_SEC / 1000);
	fprintf(stdout, "INFO: Freeing memory on CUDA device.\n");
	hipFree(cuda_matrices);
	fprintf(stdout, "INFO: computations took %d ms.\n", end);
	for (int i = 1; i < comm_size; i += 1) {
		MPI_Irecv(aes_matrices_flat + (portion * i * aes_matrix_size), (i == comm_size - 1) ? last_portion : portion, MPI_AES_MATRIX, i, 0, MPI_COMM_WORLD, &(req[i - 1]));
	}
	MPI_Barrier(MPI_COMM_WORLD);
	fprintf(stdout, "INFO: ID0 - buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", aes_matrices_flat[0], aes_matrices_flat[1], aes_matrices_flat[2], aes_matrices_flat[3], aes_matrices_flat[aes_datasize - 4], aes_matrices_flat[aes_datasize - 3], aes_matrices_flat[aes_datasize - 2], aes_matrices_flat[aes_datasize - 1]);
	return (int)end;
}

int aes_shiftrows_cuda(char* aes_matrices_flat, size_t aes_datasize) {
	fprintf(stdout, "INFO: performing AES shift rows on matrix array using NVidia CUDA.\n");
	char* cuda_matrices;
	clock_t start, end;
	aesbyte_t aes_side_len = aes_matrices_flat[0];
	fprintf(stdout, "INFO: AES matrix side length is %u.\n", aes_side_len);
	size_t aes_matrix_size = (aes_side_len * aes_side_len + 1) * sizeof(aesbyte_t);
	fprintf(stdout, "INFO: AES matrix size is %llu bytes.\n", aes_matrix_size);
    size_t aes_matrix_qty = aes_datasize / aes_matrix_size;
	if (aes_matrix_size * aes_matrix_qty != aes_datasize) {
		fprintf(stdout, "ERROR in aes_shiftrows_cuda(): expected %llu bytes, got %llu bytes.\n", aes_matrix_size * aes_matrix_qty, aes_datasize);
		return -1;
	}
	fprintf(stdout, "INFO: Received %llu AES matrices.\n", aes_matrix_qty);
	fprintf(stdout, "INFO: buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", aes_matrices_flat[0], aes_matrices_flat[1], aes_matrices_flat[2], aes_matrices_flat[3], aes_matrices_flat[aes_datasize - 4], aes_matrices_flat[aes_datasize - 3], aes_matrices_flat[aes_datasize - 2], aes_matrices_flat[aes_datasize - 1]);
	fprintf(stdout, "INFO: Allocating memory on CUDA device.\n");
	start = clock();
    hipMalloc((void**)&cuda_matrices, aes_matrix_qty * aes_matrix_size);
	fprintf(stdout, "INFO: Copying AES matrices to CUDA device.\n");
    hipMemcpy(cuda_matrices, aes_matrices_flat, aes_matrix_qty * aes_matrix_size, hipMemcpyHostToDevice);
	size_t launch_blocks, launch_threads;
	launch_blocks = aes_matrix_qty / CUDA_THREADS_PER_BLOCK + 1;
	launch_threads = CUDA_THREADS_PER_BLOCK;
	fprintf(stdout, "INFO: Launching CUDA kernel on device: %llu blocks, %llu threads.\n", launch_blocks, launch_threads);
    aes_shift_matrices_cuda <<<launch_blocks, launch_threads>>>(cuda_matrices, aes_matrix_size, aes_side_len, aes_matrix_qty);
	fprintf(stdout, "INFO: AES shift rows on matrix array completed.\n");
	fprintf(stdout, "INFO: Fetching shifted AES matrices from CUDA device.\n");
	hipMemcpy(aes_matrices_flat, cuda_matrices, aes_matrix_qty * aes_matrix_size, hipMemcpyDeviceToHost);
	end = (clock() - start) / (CLOCKS_PER_SEC / 1000);
	fprintf(stdout, "INFO: buffer overview -\n");
	fprintf(stdout, "%d %d %d %d ... %d %d %d %d\n", aes_matrices_flat[0], aes_matrices_flat[1], aes_matrices_flat[2], aes_matrices_flat[3], aes_matrices_flat[aes_datasize - 4], aes_matrices_flat[aes_datasize - 3], aes_matrices_flat[aes_datasize - 2], aes_matrices_flat[aes_datasize - 1]);
	fprintf(stdout, "INFO: Freeing memory on CUDA device.\n");
    hipFree(cuda_matrices);
	fprintf(stdout, "INFO: computations took %d ms.\n", end);
    return (end == 0) ? 1 : (int)end;
}

int main(int argc, char* argv[]) {
	MPI_Init(&argc, &argv);
	int comm_rank, comm_size;
	size_t matrix_qty;
	MPI_Comm_rank(MPI_COMM_WORLD, &comm_rank);
	MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
	if (!comm_rank) {
		if ((argc != 1) && (argc != 2)) {
			fprintf(stdout, "Usage:\nlogic_gpu.exe [CUDA_ONLY | CUDA_MPI]\nExample:\nlogic.exe\nlogic.exe CUDA_ONLY\n");
			exit(EXIT_FAILURE);
		}
		size_t recv_datasize, matrix_qty;
		SOCKET connect_socket;
		if (tcp_establish_conn_client(&connect_socket)) {
			fprintf(stderr, "ERROR in main(): general failure establishing connection with server.\n");
			exit(EXIT_FAILURE);
		}
		char* aes_matrices_flat = tcp_receive_matrices(&recv_datasize, connect_socket);
		if (!aes_matrices_flat) {
			fprintf(stderr, "ERROR in main(): general failure receiving matrices by TCP.\n");
			exit(EXIT_FAILURE);
		}
		int aes_time;
		if (argc == 1) {
			fprintf(stdout, "INFO: using NVidia CUDA.\n");
			aes_time = aes_shiftrows_cuda(aes_matrices_flat, recv_datasize);
		}
		else if (!strcmp(argv[1], "CUDA_ONLY")) {
			fprintf(stdout, "INFO: using NVidia CUDA.\n");
			aes_time = aes_shiftrows_cuda(aes_matrices_flat, recv_datasize);
		}
		else if (!strcmp(argv[1], "CUDA_MPI")) {
			fprintf(stdout, "INFO: using NVidia CUDA with MS-MPI.\n");
			aes_time = aes_shiftrows_cuda_mpi(aes_matrices_flat, recv_datasize, &matrix_qty);
		}
		else {
			fprintf(stdout, "ERROR in main(): general failure parsing command line arguments.\nUsage:\nlogic_gpu.exe [CUDA_ONLY | CUDA_MPI]\nExample:\nlogic.exe\nlogic.exe CUDA_ONLY\n");
			exit(EXIT_FAILURE);
		}
		if (aes_time < 0) {
			fprintf(stderr, "ERROR in main(): general failure shifting AES matrix array.\n");
			exit(EXIT_FAILURE);
		}
		char* tcp_data = aes_matrices_flat;
		if (!tcp_data) {
			fprintf(stderr, "ERROR in main(): general failure flattening matrices.\n");
			exit(EXIT_FAILURE);
		}
		if (tcp_send_matrices(tcp_data, recv_datasize, connect_socket)) {
			fprintf(stderr, "ERROR in main(): general failure sending matrices over TCP.\n");
			exit(EXIT_FAILURE);
		}
		free(tcp_data);
		if (tcp_send_time(aes_time, connect_socket)) {
			fprintf(stderr, "ERROR in main(): general failure sending time info over TCP.\n");
			exit(EXIT_FAILURE);
		}
		fprintf(stdout, "INFO: all done!\n");
	}
	else {
		size_t aes_meta[2];
		MPI_Status mpi_stat;
		fprintf(stdout, "INFO: ID%d - creating a contiguous MPI data type for metadata.\n", comm_rank);
		MPI_Datatype MPI_AES_META;
		MPI_Type_contiguous(2, MPI_UNSIGNED_LONG_LONG, &MPI_AES_META);
		MPI_Type_commit(&MPI_AES_META);
		fprintf(stdout, "INFO: ID%d - receiving matrix size and quantity metadata from ID0.\n", comm_rank);
		MPI_Recv(&aes_meta, 1, MPI_AES_META, 0, 0, MPI_COMM_WORLD, &mpi_stat);
		MPI_Barrier(MPI_COMM_WORLD);
		fprintf(stdout, "INFO: ID%d - received matrix size = %llu, matrix quantity = %llu from ID0.\n", comm_rank, aes_meta[0], aes_meta[1]);
		fprintf(stdout, "INFO: ID%d - creating a contiguous MPI data type for AES matrix.\n", comm_rank);
		MPI_Datatype MPI_AES_MATRIX;
		MPI_Type_contiguous(aes_meta[0], MPI_CHAR, &MPI_AES_MATRIX);
		MPI_Type_commit(&MPI_AES_MATRIX);
		fprintf(stdout, "INFO: ID%d - allocating %llu bytes for %llu matrices.\n", comm_rank, aes_meta[0] * aes_meta[1], aes_meta[1]);
		char* portion_buf = (char*)malloc(sizeof(char) * aes_meta[0] * aes_meta[1]);
		fprintf(stdout, "INFO: ID%d - receiving matrix array portion from ID0.\n", comm_rank);
		MPI_Recv(portion_buf, aes_meta[1], MPI_AES_MATRIX, 0, 0, MPI_COMM_WORLD, &mpi_stat);
		fprintf(stdout, "INFO: ID%d - received matrix array portion from ID0, %llu matrices.\n", comm_rank, aes_meta[1]);
		fprintf(stdout, "INFO: ID%d - performing AES shift on my portion.\n", comm_rank);
		aesbyte_t* temp_row = (aesbyte_t*)malloc(sizeof(aesbyte_t) * portion_buf[0]);
		for (size_t k = 0; k < aes_meta[1]; k += 1) {
			for (aesbyte_t i = 0; i < (aesbyte_t)portion_buf[0]; i += 1) {
				for (aesbyte_t j = 0; j < (aesbyte_t)portion_buf[0]; j += 1)
					temp_row[j] = portion_buf[(k * aes_meta[0] + 1) + i * portion_buf[0] + j];
				for (aesbyte_t j = 0; j < (aesbyte_t)portion_buf[0]; j += 1)
					portion_buf[(k * aes_meta[0] + 1) + i * portion_buf[0] + j] = temp_row[(i + j) % portion_buf[0]];
			}
		}
		fprintf(stdout, "INFO: ID%d - sending processed matrix array portion back to ID0.\n", comm_rank);
		MPI_Send(portion_buf, aes_meta[1], MPI_AES_MATRIX, 0, 0, MPI_COMM_WORLD);
		MPI_Barrier(MPI_COMM_WORLD);
		free(temp_row);
		free(portion_buf);
	}
	MPI_Finalize();
	exit(EXIT_SUCCESS);
}